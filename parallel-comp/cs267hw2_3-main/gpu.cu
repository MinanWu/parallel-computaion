#include "hip/hip_runtime.h"
#include "common.h"
#include <hip/hip_runtime.h>
#include <thrust/execution_policy.h>
#include <thrust/device_vector.h>
#include <thrust/scan.h>



int num_blocks;
#define threads_per_block 256

double bin_width;
int row_length;
int num_bins;

int* part_ids;
int* part_bin_indices;
int* bin_counts;
int* bin_start_indices;



__global__ void set_ax_ay_gpu(particle_t* parts, int num_parts, int* bin_start_indices, int num_bins) {
	int tid = threadIdx.x + (blockIdx.x * blockDim.x);
	if (tid >= num_parts) {
		return;
	}
	bin_start_indices[num_bins] = num_parts;
	parts[tid].ax = 0;
	parts[tid].ay = 0;
}

void init_simulation(particle_t* parts, int num_parts, double size) {
	num_blocks = ((num_parts + (threads_per_block - 1)) / threads_per_block);
	bin_width = (2 * cutoff);
	row_length = std::ceil(size / bin_width);
	num_bins = pow(row_length, 2);
	hipMalloc(((void**) &part_ids), (num_parts * sizeof(int)));
	hipMalloc(((void**) &part_bin_indices), (num_parts * sizeof(int)));
	hipMalloc(((void**) &bin_counts), (num_bins * sizeof(int)));
	hipMalloc(((void**) &bin_start_indices), ((num_bins + 1) * sizeof(int)));
	set_ax_ay_gpu<<<num_blocks, threads_per_block>>>(parts, num_parts, bin_start_indices, num_bins);
}



__device__ void apply_force_gpu(particle_t& particle, particle_t& neighbor) {
	double dx = neighbor.x - particle.x;
	double dy = neighbor.y - particle.y;
	double r2 = (dx * dx) + (dy * dy);
	if (r2 > cutoff * cutoff) {
		return;
	}
	// r2 = fmax( r2, min_r*min_r );
	r2 = ((r2 > min_r * min_r) ? (r2) : (min_r * min_r));
	double r = sqrt(r2);
	double coef = (1 - cutoff / r) / r2 / mass;
	// To consider: How does cutting down computations compare to adding these barriers?
	atomicAdd(&(particle.ax), (coef * dx));
	atomicAdd(&(particle.ay), (coef * dy));
}

__device__ void apply_force_symmetric_gpu(particle_t& particle, particle_t& neighbor) {
	double dx = neighbor.x - particle.x;
	double dy = neighbor.y - particle.y;
	double r2 = (dx * dx) + (dy * dy);
	if (r2 > cutoff * cutoff) {
		return;
	}
	// r2 = fmax( r2, min_r*min_r );
	r2 = ((r2 > min_r * min_r) ? (r2) : (min_r * min_r));
	double r = sqrt(r2);
	double coef = (1 - cutoff / r) / r2 / mass;
	atomicAdd(&(particle.ax), (coef * dx));
	atomicAdd(&(particle.ay), (coef * dy));
	atomicAdd(&(neighbor.ax), -(coef * dx));
	atomicAdd(&(neighbor.ay), -(coef * dy));
}


__global__ void compute_forces_gpu(particle_t* parts, int num_parts, double bin_width, int row_length, int* bin_counts, int* bin_start_indices, int* part_ids, int* part_bin_indices) {

	int tid = threadIdx.x + (blockIdx.x * blockDim.x);
	if (tid >= num_parts) {
		return;
	}

	int bin_index = part_bin_indices[tid];
	int row_index = bin_index / row_length;
	int column_index = bin_index % row_length;

	int bin_start_index = bin_start_indices[bin_index];
	int next_bin_start_index = bin_start_indices[bin_index + 1];
	for (int j = bin_start_index; ((j < next_bin_start_index) && (part_ids[j] != tid)); j += 1) {
		apply_force_symmetric_gpu(
			parts[tid],
			parts[part_ids[j]]
		);
	}

	if (column_index <= (row_length - 2)) {
		int mid_right_index = bin_index + 1;
		int bin_start_index = bin_start_indices[mid_right_index];
		int next_bin_start_index = bin_start_indices[mid_right_index + 1];
		for (int j = bin_start_index; j < next_bin_start_index; j += 1) {
			apply_force_symmetric_gpu(
				parts[tid],
				parts[part_ids[j]]
			);
		}
		if (row_index >= 1) {
			int top_right_index = bin_index - row_length + 1;
			int bin_start_index = bin_start_indices[top_right_index];
			int next_bin_start_index = bin_start_indices[top_right_index + 1];
			for (int j = bin_start_index; j < next_bin_start_index; j += 1) {
				apply_force_symmetric_gpu(
					parts[tid],
					parts[part_ids[j]]
				);
			}
		}
	}

	if (row_index <= (row_length - 2)) {
		int bottom_center_index = bin_index + row_length;
		int bin_start_index = bin_start_indices[bottom_center_index];
		int next_bin_start_index = bin_start_indices[bottom_center_index + 1];
		for (int j = bin_start_index; j < next_bin_start_index; j += 1) {
			apply_force_symmetric_gpu(
				parts[tid],
				parts[part_ids[j]]
			);
		}
		if (column_index <= (row_length - 2)) {
			int bottom_right_index = bin_index + row_length + 1;
			int bin_start_index = bin_start_indices[bottom_right_index];
			int next_bin_start_index = bin_start_indices[bottom_right_index + 1];
			for (int j = bin_start_index; j < next_bin_start_index; j += 1) {
				apply_force_symmetric_gpu(
					parts[tid],
					parts[part_ids[j]]
				);
			}
		}
	}

}


__global__ void move_gpu(particle_t* parts, int num_parts, double size) {
	int tid = threadIdx.x + (blockIdx.x * blockDim.x);
	if (tid >= num_parts) {
		return;
	}
	particle_t* p = &(parts[tid]);
	p->vx += (p->ax * dt);
	p->vy += (p->ay * dt);
	p->x += (p->vx * dt);
	p->y += (p->vy * dt);
	// Bounce from walls
	while (p->x < 0 || p->x > size) {
		p->x = p->x < 0 ? -(p->x) : 2 * size - p->x;
		p->vx = -(p->vx);
	}
	while (p->y < 0 || p->y > size) {
		p->y = p->y < 0 ? -(p->y) : 2 * size - p->y;
		p->vy = -(p->vy);
	}
	parts[tid].ax = 0;
	parts[tid].ay = 0;
}


__global__ void count_particles_gpu(particle_t* parts, int num_parts, double bin_width, int row_length, int* bin_counts, int* part_bin_indices) {
	int tid = threadIdx.x + (blockIdx.x * blockDim.x);
	if (tid >= num_parts) {
		return;
	}
	int bin_index = (
		 (floor(parts[tid].x / bin_width) * row_length)
		+ floor(parts[tid].y / bin_width)
	);
	atomicAdd(&(bin_counts[bin_index]), 1);
	part_bin_indices[tid] = bin_index;
}

__global__ void arrange_part_ids_gpu(particle_t* parts, int num_parts, double bin_width, int row_length, int* bin_counts, int* bin_start_indices, int* part_ids, int* part_bin_indices) {
	int tid = threadIdx.x + (blockIdx.x * blockDim.x);
	if (tid >= num_parts) {
		return;
	}
	int bin_index = part_bin_indices[tid];
	int next_subindex = atomicAdd(&(bin_counts[bin_index]), -1);
	part_ids[(bin_start_indices[bin_index] + next_subindex - 1)] = tid;
}



void simulate_one_step(particle_t* parts, int num_parts, double size) {

	// Compute forces
	compute_forces_gpu<<<num_blocks, threads_per_block>>>(
		parts,
		num_parts,
		bin_width,
		row_length,
		bin_counts,
		bin_start_indices,
		part_ids,
		part_bin_indices
	);

	// Move particles
	move_gpu<<<num_blocks, threads_per_block>>>(parts, num_parts, size);

	// Re-bin particles
	hipMemset(bin_counts, 0, (num_bins * sizeof(int)));
	count_particles_gpu<<<num_blocks, threads_per_block>>>(
		parts,
		num_parts,
		bin_width,
		row_length,
		bin_counts,
		part_bin_indices
	);
	thrust::exclusive_scan(
		thrust::device,
		bin_counts,
		(bin_counts + num_bins),
		bin_start_indices
	);
	arrange_part_ids_gpu<<<num_blocks, threads_per_block>>>(
		parts,
		num_parts,
		bin_width,
		row_length,
		bin_counts,
		bin_start_indices,
		part_ids,
		part_bin_indices
	);

}
